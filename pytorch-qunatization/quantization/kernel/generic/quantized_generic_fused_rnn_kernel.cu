
#include <hip/hip_runtime.h>
/* 
Copyright (c) 2018-     Xilinx, Inc              (Alessandro Pappalardo)
Copyright (c) 2016-     Facebook, Inc            (Adam Paszke)
Copyright (c) 2014-     Facebook, Inc            (Soumith Chintala)
Copyright (c) 2011-2014 Idiap Research Institute (Ronan Collobert)
Copyright (c) 2012-2014 Deepmind Technologies    (Koray Kavukcuoglu)
Copyright (c) 2011-2012 NEC Laboratories America (Koray Kavukcuoglu)
Copyright (c) 2011-2013 NYU                      (Clement Farabet)
Copyright (c) 2006-2010 NEC Laboratories America (Ronan Collobert, Leon Bottou, Iain Melvin, Jason Weston)
Copyright (c) 2006      Idiap Research Institute (Samy Bengio)
Copyright (c) 2001-2004 Idiap Research Institute (Ronan Collobert, Samy Bengio, Johnny Mariethoz)

All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright
   notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
   notice, this list of conditions and the following disclaimer in the
   documentation and/or other materials provided with the distribution.

3. Neither the names of Xilinx, Facebook, Deepmind Technologies, NYU, 
   NEC Laboratories America and IDIAP Research Institute nor the names 
   of its contributors may be used to endorse or promote products derived 
   from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE. 
*/

#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/quantized_generic_fused_rnn_kernel.cu"
#else

#include <cstdarg>
#include <quantized_generic_fused_rnn_kernel_helper.cuh>
#include <quantized_generic_fused_rnn_kernel_impl.cuh>
#include <quantized_generic_fused_rnn_kernel_internal_wrap.cuh>

void THNN_(QuantizedLSTMFused_updateOutput)(
   THCState *state,
   THCTensor *input,
   THCTensor *hidden,
   THCTensor *bias1,
   THCTensor *bias2,
   THCTensor *cx,
   THCTensor *hy,
   THCTensor *cy,
   THCTensor *quantizationBitWidth)
{
  THCTensor_(resizeAs)(state, hy, cx);
  THCTensor_(resizeAs)(state, cy, cx);
  THNN_(FusedRNNAssertSizes)(state, 4, 5, input, hidden, hy, cy, cx);

  bool canUse32bi = THNN_(canUse32BitIndexMath)
      (state, 7, input, hidden, bias1, bias2, hy, cy, cx);
  

  if (canUse32bi) {
    THNN_(QuantizedLSTM_forw_ind_wrap)<uint32_t>
      (state, input, hidden, bias1, bias2, cx, hy, cy, quantizationBitWidth);
  } else {
    THNN_(QuantizedLSTM_forw_ind_wrap)<uint64_t>
      (state, input, hidden, bias1, bias2, cx, hy, cy, quantizationBitWidth);
  }
  THCudaCheck(cudaGetLastError());
}

void THNN_(QuantizedLSTMFusedNoBias_updateOutput)(
   THCState *state,
   THCTensor *input,
   THCTensor *hidden,
   THCTensor *cx,
   THCTensor *hy,
   THCTensor *cy,
   THCTensor *quantizationBitWidth)
{
  THCTensor_(resizeAs)(state, hy, cx);
  THCTensor_(resizeAs)(state, cy, cx);
  THNN_(FusedRNNAssertSizes)(state, 4, 5, input, hidden, hy, cy, cx);

  bool canUse32bi = THNN_(canUse32BitIndexMath)
      (state, 5, input, hidden, hy, cy, cx);
  
  if(canUse32bi){
    THNN_(QuantizedLSTM_forw_ind_wrap)<uint32_t>
      (state, input, hidden, NULL, NULL, cx, hy, cy, quantizationBitWidth);
  } else {
    THNN_(QuantizedLSTM_forw_ind_wrap)<uint64_t>
      (state, input, hidden, NULL, NULL, cx, hy, cy, quantizationBitWidth);
  }
  THCudaCheck(cudaGetLastError());
}

void THNN_(QuantizedLSTMFused_updateGradInput)(
   THCState *state,
   THCTensor *storage,
   THCTensor *gradInGates,
   THCTensor *cx,
   THCTensor *cy,
   THCTensor *gradOutput,
   THCTensor *gradOutputCell,
   THCTensor *gradInputCx)
{
  THCTensor_(resizeAs)(state, gradInputCx, gradOutput);
  THCUNN_assertSameGPU(state, 7, storage, gradInGates, cx, cy,
               gradOutput, gradOutputCell, gradInputCx);
  THNN_(FusedRNNAssertSizes)
    (state, 4, 7, storage, gradInGates, cx, cy,
     gradOutput, gradOutputCell, gradInputCx);

  bool canUse32bi = THNN_(canUse32BitIndexMath)
    (state, 7, storage, gradInGates, cx, cy,
     gradOutput, gradOutputCell, gradInputCx);

  if(canUse32bi){
    THNN_(QuantizedLSTM_back_ind_wrap)<uint32_t>
      (state, storage, gradInGates, cx, cy,
       gradOutput, gradOutputCell, gradInputCx);
  }else{
    THNN_(QuantizedLSTM_back_ind_wrap)<uint64_t>
      (state, storage, gradInGates, cx, cy,
       gradOutput, gradOutputCell, gradInputCx);
  }
  THCudaCheck(cudaGetLastError());
}

void THNN_(QuantizedGRUFused_updateOutput)(
   THCState *state,
   THCTensor *input,
   THCTensor *hidden,
   THCTensor *bias1,
   THCTensor *bias2,
   THCTensor *hx,
   THCTensor *hy,
   THCTensor *storage)
{
  THCTensor_(resizeAs)(state, hy, hx);
  THNN_(FusedRNNAssertSizes)(state, 3, 4, input, hidden, hx, hy);
  THArgCheck(THCTensor_(nElement)(state, storage) ==
             THCTensor_(nElement)(state, hx)*5,
             3, "Storage tensor for fused kernel was not sized correctly.");

  bool canUse32bi = THNN_(canUse32BitIndexMath)
      (state, 7, input, hidden, hx, hy, bias1, bias2, storage);

  if (canUse32bi) {
    THNN_(QuantizedGRU_forw_ind_wrap)<uint32_t>
      (state, input, hidden, bias1, bias2, hx, hy, storage);
  } else {
    THNN_(QuantizedGRU_forw_ind_wrap)<uint64_t>
      (state, input, hidden, bias1, bias2, hx, hy, storage);
  }
  THCudaCheck(cudaGetLastError());
}

void THNN_(QuantizedGRUFusedNoBias_updateOutput)(
   THCState *state,
   THCTensor *input,
   THCTensor *hidden,
   THCTensor *hx,
   THCTensor *hy,
   THCTensor *storage)
{
  THCTensor_(resizeAs)(state, hy, hx);
  THNN_(FusedRNNAssertSizes)(state, 3, 4, input, hidden, hx, hy);
  THArgCheck(THCTensor_(nElement)(state, storage) ==
             THCTensor_(nElement)(state, hx)*5,
             3, "Storage tensor for fused kernel was not sized correctly.");

  bool canUse32bi = THNN_(canUse32BitIndexMath)
      (state, 5, input, hidden, hx, hy, storage);
  
  if (canUse32bi) {
    THNN_(QuantizedGRU_forw_ind_wrap)<uint32_t>
      (state, input, hidden, NULL, NULL, hx, hy, storage);
  } else {
    THNN_(QuantizedGRU_forw_ind_wrap)<uint64_t>
      (state, input, hidden, NULL, NULL, hx, hy, storage);
  }
  THCudaCheck(cudaGetLastError());
}

void THNN_(QuantizedGRUFused_updateGradInput)(
   THCState *state,
   THCTensor *gradInInput,
   THCTensor *gradInHidden,
   THCTensor *gradOutput,
   THCTensor *gradInputHx,
   THCTensor *storage)
{
  THCTensor_(resizeAs)(state, gradInputHx, gradOutput);
  THCUNN_assertSameGPU(state, 5, gradInInput, gradInHidden, gradOutput, gradInputHx, storage);
  THNN_(FusedRNNAssertSizes)(state, 3, 4, gradInInput, gradInHidden, gradOutput, gradInputHx);
  bool canUse32bi = THNN_(canUse32BitIndexMath)(state, 5, gradInInput, gradInHidden,
                                                gradOutput, gradInputHx, storage);
  if(canUse32bi){
    THNN_(QuantizedGRU_back_ind_wrap)<uint32_t>
      (state, gradInInput, gradInHidden, gradOutput, gradInputHx, storage);
  }else{
    THNN_(QuantizedGRU_back_ind_wrap)<uint64_t>
      (state, gradInInput, gradInHidden, gradOutput, gradInputHx, storage);
  }

  THCudaCheck(cudaGetLastError());
}

#include <quantized_fused_rnn_kernel_cleanup.h>

#endif
